#include "hip/hip_runtime.h"
// Writer: Junhyuck Woo
// Lecture: Multicore Computing
// Organization: Chung-Ang University
// Deadline: June3 20, 2020
// Project #4
// - Thrust

#include <thrust/fill.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <time.h>

using namespace std;
#define N 2000000.0

int main(int argc, char* argv[])
{
    float sum = 0;
    clock_t start_time, end_time;
    double exec_time = 0;
    
    // allocate three device_vectors
    thrust::device_vector<float> X(N);
    thrust::device_vector<float> Y(N);
    thrust::device_vector<float> Z(N);

    // Start timer;
    start_time = clock();

    // initialize X to 0,1,2,3, ....
    thrust::sequence(X.begin(), X.end());
    thrust::fill(Y.begin(), Y.end(), N);
    // Divide X as N
    thrust::transform(X.begin(), X.end(), Y.begin(), X.begin(), thrust::divides<float>());

    // Calculation
    thrust::transform(X.begin(), X.end(), X.begin(), X.begin(), thrust::multiplies<float>());// X = X*X 
    thrust::fill(Y.begin(), Y.end(), 1.0); // Y <- 1.0
    thrust::transform(X.begin(), X.end(), Y.begin(), X.begin(), thrust::plus<float>()); // X = X + 1
    thrust::fill(Y.begin(), Y.end(), 4.0); // Y <- 4.0
    thrust::transform(Y.begin(), Y.end(), X.begin(), Z.begin(), thrust::divides<float>()); // z = 4.0 / X
    thrust::fill(Y.begin(), Y.end(), N);
    thrust::transform(Z.begin(), Z.end(), Y.begin(), Z.begin(), thrust::divides<float>()); // z = z / N

    // Sum the calculation result
    sum = thrust::reduce(Z.begin(), Z.end(), (float)0.0, thrust::plus<float>());

    // End timer
    end_time = clock();
    exec_time = (double)(end_time - start_time)*1000 / CLOCKS_PER_SEC;

    // Print the result
    cout << "N: 2000000.0" << endl;
    cout << "Excution Time: " << exec_time << " ms" << endl;
    cout << "Result: " << sum << endl;

    return 0;
}