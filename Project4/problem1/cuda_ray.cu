
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define CUDA 0
#define OPENMP 1
#define SPHERES 20

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

struct Sphere {
    float   r,b,g;
    float   radius;
    float   x,y,z;
};

__device__ float hit( float x, float y, float z, float ox, float oy, float *n,  float radius ) {
    float dx = ox - x;
    float dy = oy - y;
    if (dx*dx + dy*dy < radius*radius) {
        float dz = sqrtf( radius*radius - dx*dx - dy*dy );
        *n = dz / sqrtf( radius * radius );
        return dz + z;
    }
    return -INF;
}

__global__ void kernel(struct Sphere* s, unsigned char* ptr)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y*DIM;
    float ox = (x - DIM/2);
    float oy = (y - DIM/2);

    float r=0, g=0, b=0;
    float   maxz = -INF;
    for(int i=0; i<SPHERES; i++) {
        float   n;
        float   t = hit( s[i].x, s[i].y, s[i].z, ox, oy, &n, s[i].radius );
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset*4 + 0] = (int)(r * 255);
    ptr[offset*4 + 1] = (int)(g * 255);
    ptr[offset*4 + 2] = (int)(b * 255);
    ptr[offset*4 + 3] = 255;
}

void ppm_write(unsigned char* bitmap, int xdim,int ydim, FILE* fp)
{
    int i,x,y;
    fprintf(fp,"P3\n");
    fprintf(fp,"%d %d\n",xdim, ydim);
    fprintf(fp,"255\n");
    for (y=0;y<ydim;y++) {
        for (x=0;x<xdim;x++) {
            i=x+y*xdim;
            fprintf(fp,"%d %d %d ",bitmap[4*i],bitmap[4*i+1],bitmap[4*i+2]);
        }
        fprintf(fp,"\n");
    }
}

int main(int argc, char* argv[])
{
    double exe_time;
    clock_t start_time, end_time;
    struct Sphere *temp_s;
    unsigned char* bitmap;
    struct Sphere *d_temp_s;
    unsigned char* d_bitmap;
    dim3 blocks(DIM,DIM,1);

    // Error detection code
    if (argc!=2) {
        printf("> a.out [filename.ppm]\n");
        printf("for example, '> a.out result.ppm' means executing CUDA\n");
        exit(0);
    }

    // Start Timer
    srand(time(NULL));
    start_time = clock();

    // Allocate the memory on host
    bitmap = (unsigned char*)malloc(sizeof(unsigned char)*DIM*DIM*4);
    temp_s = (struct Sphere*)malloc(sizeof(struct Sphere) * SPHERES);
    // Allocate the memory on device
    hipMalloc( (void**)&d_temp_s, sizeof(struct Sphere) * SPHERES );
    hipMalloc( (void**)&d_bitmap, sizeof(unsigned char)*DIM*DIM*4 );

    // Generate the spheres
    for (int i=0; i<SPHERES; i++) {
        temp_s[i].r = rnd( 1.0f );
        temp_s[i].g = rnd( 1.0f );
        temp_s[i].b = rnd( 1.0f );
        temp_s[i].x = rnd( 2000.0f ) - 1000;
        temp_s[i].y = rnd( 2000.0f ) - 1000;
        temp_s[i].z = rnd( 2000.0f ) - 1000;
        temp_s[i].radius = rnd( 200.0f ) + 40;
    }

    //  Move data to device
    hipMemcpy ( d_temp_s, temp_s, sizeof(struct Sphere) * SPHERES, hipMemcpyHostToDevice );

    // Calculate the ray
    kernel<<<blocks, 1>>>(d_temp_s, d_bitmap);
    hipDeviceSynchronize();
    hipMemcpy ( bitmap, d_bitmap, sizeof(unsigned char)*DIM*DIM*4, hipMemcpyDeviceToHost );

    // open the file
    FILE* fp = fopen(argv[1],"w");
    ppm_write(bitmap,DIM,DIM,fp);   // Write the image

    // Stop Timer
    end_time = clock();
    exe_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;

    // Print the result
    printf("CUDA ray tracing: %f sec\n", exe_time);
    printf("[%s] was generated\n", argv[1]);

    // Close the file and free the memory
    fclose(fp);
    free(bitmap);
    free(temp_s);
    hipFree(d_bitmap);
    hipFree(d_temp_s);
    return 0;
}